#include "hip/hip_runtime.h"
#include <cmath>
#include <cstddef>
#include <cstdint>
#include "kernel.h"
#include "tiff_image.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include "check_cuda_errors.h"

const uint8_t kBlockSize = 16;

__constant__ int d_kernel_sobel[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__constant__ int d_kernel_prewitt[9] = {-1, 0, 1, -1, 0, 1, -1, 0, 1};
__constant__ int d_kernel_sobel_sep[3] = {1, 2, 1};
__constant__ int d_kernel_prewitt_sep[3] = {1, 1, 1};
__constant__ int d_kernel_gradient[3] = {-1, 0, 1};

bool CheckFreeMem(size_t required_memory) {
  size_t free_memory, total_memory;
  hipFree(nullptr);
  checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
  return free_memory > required_memory;
}

__device__ __forceinline__ int Clamp(int val, int min_val, int max_val) {
  return max(min_val, min(val, max_val));
}

/**
 * @brief Применяет пользовательское ядро к входному изображению.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param dst Указатель на результирующее изображение на устройстве.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 * @param kernel Указатель на ядро на устройстве.
 * @param ksize Размер ядра (предполагается квадратное).
 */
__global__ void CudaSetKernel(uint16_t* src, uint16_t* dst, size_t height,
                              size_t width, int* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int g = 0;
    for (int k = 0; k < ksize; k++) {
      for (int l = 0; l < ksize; l++) {
        int x = j + l - ksize / 2;
        int y = i + k - ksize / 2;
        x = Clamp(x, 0, width - 1);
        y = Clamp(y, 0, height - 1);
        g += src[y * width + x] * kernel[k * ksize + l];
      }
    }
    dst[i * width + j] = min(abs(g), 65535);
  }
}

__global__ void CudaSetKernelRotate(uint16_t* src, uint16_t* dst, size_t height,
                                    size_t width, int* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
    for (int k = 0; k < ksize; k++) {
      for (int l = 0; l < ksize; l++) {
        int x = j + l - ksize / 2;
        int y = i + k - ksize / 2;
        x = Clamp(x, 0, width - 1);
        y = Clamp(y, 0, height - 1);
        g_x += src[y * width + x] * kernel[k * ksize + l];
        g_y += src[y * width + x] * kernel[(ksize - 1 - l) * ksize + k];
      }
    }
    dst[i * width + j] = min(abs(g_x) + abs(g_y), 65535);
  }
}

__global__ void CudaSetKernelShared(uint16_t* src, uint16_t* dst, size_t height,
                                    size_t width, int* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ uint16_t s_tile[];
  const int radius = static_cast<int>(ksize) / 2;
  const int tile_r = kBlockSize + 2 * radius;
  const int local_x = threadIdx.x;
  const int local_y = threadIdx.y;
  const int li = Clamp(i - radius, 0, static_cast<int>(height) - 1);
  const int lj = Clamp(j - radius, 0, static_cast<int>(width) - 1);
  s_tile[local_y * tile_r + local_x] =
      src[static_cast<size_t>(li) * width + lj];
  if (local_y < 2 * radius) {
    int gy = min(i - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(height) - 1);
    s_tile[(local_y + kBlockSize) * tile_r + local_x] =
        src[static_cast<size_t>(gy) * width + lj];
  }
  if (local_x < 2 * radius) {
    int gx = min(j - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(width) - 1);
    s_tile[local_y * tile_r + (local_x + kBlockSize)] =
        src[static_cast<size_t>(li) * width + gx];
  }
  if (local_y >= static_cast<int>(kBlockSize) - 2 * radius &&
      local_x >= static_cast<int>(kBlockSize) - 2 * radius) {
    int gy = min(i + radius, static_cast<int>(height) - 1);
    int gx = min(j + radius, static_cast<int>(width) - 1);
    s_tile[(local_y + 2 * radius) * tile_r + (local_x + 2 * radius)] =
        src[static_cast<size_t>(gy) * width + gx];
  }
  __syncthreads();
  if (i < height && j < width) {
    int g = 0;
    const int k_int = static_cast<int>(ksize);
    for (int k = 0; k < k_int; ++k) {
      for (int l = 0; l < k_int; ++l) {
        int val = s_tile[(threadIdx.y + k) * tile_r + (threadIdx.x + l)];
        g += val * kernel[k * k_int + l];
      }
    }
    dst[i * width + j] = min(abs(g), 65535);
  }
}

__global__ void CudaSetKernelRotateShared(uint16_t* src, uint16_t* dst,
                                          size_t height, size_t width,
                                          int* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ uint16_t s_tile[];
  const int radius = static_cast<int>(ksize) / 2;
  const int tile_r = kBlockSize + 2 * radius;
  const int local_x = threadIdx.x;
  const int local_y = threadIdx.y;
  const int li = Clamp(i - radius, 0, static_cast<int>(height) - 1);
  const int lj = Clamp(j - radius, 0, static_cast<int>(width) - 1);
  s_tile[local_y * tile_r + local_x] =
      src[static_cast<size_t>(li) * width + lj];
  if (local_y < 2 * radius) {
    int gy = min(i - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(height) - 1);
    s_tile[(local_y + kBlockSize) * tile_r + local_x] =
        src[static_cast<size_t>(gy) * width + lj];
  }
  if (local_x < 2 * radius) {
    int gx = min(j - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(width) - 1);
    s_tile[local_y * tile_r + (local_x + kBlockSize)] =
        src[static_cast<size_t>(li) * width + gx];
  }
  if (local_y >= static_cast<int>(kBlockSize) - 2 * radius &&
      local_x >= static_cast<int>(kBlockSize) - 2 * radius) {
    int gy = min(i + radius, static_cast<int>(height) - 1);
    int gx = min(j + radius, static_cast<int>(width) - 1);
    s_tile[(local_y + 2 * radius) * tile_r + (local_x + 2 * radius)] =
        src[static_cast<size_t>(gy) * width + gx];
  }
  __syncthreads();
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
    const int k_int = static_cast<int>(ksize);
    for (int k = 0; k < k_int; ++k) {
      for (int l = 0; l < k_int; ++l) {
        int val = s_tile[(threadIdx.y + k) * tile_r + (threadIdx.x + l)];
        g_x += val * kernel[k * k_int + l];
        g_y += val * kernel[(k_int - 1 - l) * k_int + k];
      }
    }
    dst[i * width + j] = min(abs(g_x) + abs(g_y), 65535);
  }
}

/**
 * @brief Применяет ядро Собеля для вычисления границ.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param dst Указатель на результирующее изображение на устройстве.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 */
__global__ void CudaSetSobelKernel(uint16_t* src, uint16_t* dst, size_t height,
                                   size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
#pragma unroll
      for (int l = 0; l < 3; l++) {
        int x = j + l - 1;
        int y = i + k - 1;
        x = Clamp(x, 0, width - 1);
        y = Clamp(y, 0, height - 1);
        g_x += src[y * width + x] * d_kernel_sobel[k * 3 + l];
        g_y += src[y * width + x] * d_kernel_sobel[(3 - 1 - l) * 3 + k];
      }
    }
    dst[i * width + j] = min(abs(g_x) + abs(g_y), 65535);
  }
}

__global__ void CudaSetSobelKernelShared(uint16_t* src, uint16_t* dst,
                                         size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  __shared__ uint16_t s_tile[(kBlockSize + 2) * (kBlockSize + 2)];
  int li = Clamp(i - 1, 0, height - 1);
  int lj = Clamp(j - 1, 0, width - 1);
  s_tile[local_y * (kBlockSize + 2) + local_x] = src[li * width + lj];
  if (local_y < 2) {
    s_tile[(local_y + kBlockSize) * (kBlockSize + 2) + local_x] =
        src[min(i - 1 + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x < 2) {
    s_tile[local_y * (kBlockSize + 2) + (local_x + kBlockSize)] =
        src[li * width + min(j - 1 + kBlockSize, (int)width - 1)];
  }
  if (local_y >= kBlockSize - 2 && local_x >= kBlockSize - 2) {
    s_tile[(local_y + 2) * (kBlockSize + 2) + (local_x + 2)] =
        src[min(i - 1 + 2, (int)height - 1) * width +
            min(j - 1 + 2, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
#pragma unroll
      for (int l = 0; l < 3; l++) {
        int val = s_tile[(local_y + k) * (kBlockSize + 2) + (local_x + l)];
        g_x += val * d_kernel_sobel[k * 3 + l];
        g_y += val * d_kernel_sobel[(2 - l) * 3 + k];
      }
    }
    dst[i * width + j] = min(abs(g_x) + abs(g_y), 65535);
  }
}

/**
 * @brief Применяет ядро Собеля в раздельной форме для сглаживания.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param g_x Указатель на градиент по оси X.
 * @param g_y Указатель на градиент по оси Y.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 */
__global__ void CudaSetSobelKernelSmooth(uint16_t* src, int* g_x, int* g_y,
                                         size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
      int x = j + k - 1;
      int y = i + k - 1;
      x = Clamp(x, 0, width - 1);
      y = Clamp(y, 0, height - 1);
      sum_x += src[i * width + x] * d_kernel_sobel_sep[k];
      sum_y += src[y * width + j] * d_kernel_sobel_sep[k];
    }
    g_x[i * width + j] = sum_x;
    g_y[i * width + j] = sum_y;
  }
}

__global__ void CudaSetSobelKernelSmoothShared(uint16_t* src, int* g_x,
                                               int* g_y, size_t height,
                                               size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  __shared__ uint16_t s_tile[(kBlockSize + 2) * (kBlockSize + 2)];
  int li = Clamp(i - 1, 0, height - 1);
  int lj = Clamp(j - 1, 0, width - 1);
  s_tile[local_y * (kBlockSize + 2) + local_x] = src[li * width + lj];
  if (local_y < 2) {
    s_tile[(local_y + kBlockSize) * (kBlockSize + 2) + local_x] =
        src[min(i - 1 + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x < 2) {
    s_tile[local_y * (kBlockSize + 2) + (local_x + kBlockSize)] =
        src[li * width + min(j - 1 + kBlockSize, (int)width - 1)];
  }
  if (local_y >= kBlockSize - 2 && local_x >= kBlockSize - 2) {
    s_tile[(local_y + 2) * (kBlockSize + 2) + (local_x + 2)] =
        src[min(i - 1 + 2, (int)height - 1) * width +
            min(j - 1 + 2, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; ++k) {
      sum_x += s_tile[(local_y + 1) * (kBlockSize + 2) + (local_x + k)] *
               d_kernel_sobel_sep[k];
      sum_y += s_tile[(local_y + k) * (kBlockSize + 2) + (local_x + 1)] *
               d_kernel_sobel_sep[k];
    }
    g_x[i * width + j] = sum_x;
    g_y[i * width + j] = sum_y;
  }
}

/**
 * @brief Применяет ядро Превитта для вычисления границ.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param dst Указатель на результирующее изображение на устройстве.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 */
__global__ void CudaSetPrewittKernel(uint16_t* src, uint16_t* dst,
                                     size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
#pragma unroll
      for (int l = 0; l < 3; l++) {
        int x = j + l - 1;
        int y = i + k - 1;
        x = Clamp(x, 0, width - 1);
        y = Clamp(y, 0, height - 1);
        g_x += src[y * width + x] * d_kernel_prewitt[k * 3 + l];
        g_y += src[y * width + x] * d_kernel_prewitt[(3 - 1 - l) * 3 + k];
      }
    }
    dst[i * width + j] = Clamp(abs(g_x) + abs(g_y), 0, 65535);
  }
}

__global__ void CudaSetPrewittKernelShared(uint16_t* src, uint16_t* dst,
                                           size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  __shared__ uint16_t s_tile[(kBlockSize + 2) * (kBlockSize + 2)];
  int li = Clamp(i - 1, 0, height - 1);
  int lj = Clamp(j - 1, 0, width - 1);
  s_tile[local_y * (kBlockSize + 2) + local_x] = src[li * width + lj];
  if (local_y < 2) {
    s_tile[(local_y + kBlockSize) * (kBlockSize + 2) + local_x] =
        src[min(i - 1 + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x < 2) {
    s_tile[local_y * (kBlockSize + 2) + (local_x + kBlockSize)] =
        src[li * width + min(j - 1 + kBlockSize, (int)width - 1)];
  }
  if (local_y >= kBlockSize - 2 && local_x >= kBlockSize - 2) {
    s_tile[(local_y + 2) * (kBlockSize + 2) + (local_x + 2)] =
        src[min(i - 1 + 2, (int)height - 1) * width +
            min(j - 1 + 2, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int g_x = 0, g_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
#pragma unroll
      for (int l = 0; l < 3; l++) {
        int val = s_tile[(local_y + k) * (kBlockSize + 2) + (local_x + l)];
        g_x += val * d_kernel_prewitt[k * 3 + l];
        g_y += val * d_kernel_prewitt[(2 - l) * 3 + k];
      }
    }
    dst[i * width + j] = min(abs(g_x) + abs(g_y), 65535);
  }
}

/**
 * @brief Применяет ядро Превитта в раздельной форме для усреднения.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param g_x Указатель на градиент по оси X.
 * @param g_y Указатель на градиент по оси Y.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 */
__global__ void CudaSetPrewittKernelAverage(uint16_t* src, int* g_x, int* g_y,
                                            size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
      int x = j + k - 1;
      int y = i + k - 1;
      x = Clamp(x, 0, width - 1);
      y = Clamp(y, 0, height - 1);
      sum_x += src[i * width + x] * d_kernel_prewitt_sep[k];
      sum_y += src[y * width + j] * d_kernel_prewitt_sep[k];
    }
    g_x[i * width + j] = sum_x;
    g_y[i * width + j] = sum_y;
  }
}

__global__ void CudaSetPrewittKernelAverageShared(uint16_t* src, int* g_x,
                                                  int* g_y, size_t height,
                                                  size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  __shared__ uint16_t s_tile[(kBlockSize + 2) * (kBlockSize + 2)];
  int li = Clamp(i - 1, 0, height - 1);
  int lj = Clamp(j - 1, 0, width - 1);
  s_tile[local_y * (kBlockSize + 2) + local_x] = src[li * width + lj];
  if (local_y < 2) {
    s_tile[(local_y + kBlockSize) * (kBlockSize + 2) + local_x] =
        src[min(i - 1 + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x < 2) {
    s_tile[local_y * (kBlockSize + 2) + (local_x + kBlockSize)] =
        src[li * width + min(j - 1 + kBlockSize, (int)width - 1)];
  }
  if (local_y >= kBlockSize - 2 && local_x >= kBlockSize - 2) {
    s_tile[(local_y + 2) * (kBlockSize + 2) + (local_x + 2)] =
        src[min(i - 1 + 2, (int)height - 1) * width +
            min(j - 1 + 2, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; ++k) {
      sum_x += s_tile[(local_y + 1) * (kBlockSize + 2) + (local_x + k)] *
               d_kernel_prewitt_sep[k];
      sum_y += s_tile[(local_y + k) * (kBlockSize + 2) + (local_x + 1)] *
               d_kernel_prewitt_sep[k];
    }
    g_x[i * width + j] = sum_x;
    g_y[i * width + j] = sum_y;
  }
}

/**
 * @brief Вычисляет разность градиентов с использованием раздельного ядра.
 *
 * @param g_x Указатель на градиент по оси X.
 * @param g_y Указатель на градиент по оси Y.
 * @param result_x Указатель на результат по оси X.
 * @param result_y Указатель на результат по оси Y.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 */
__global__ void CudaSepKernelDiff(int* g_x, int* g_y, int* result_x,
                                  int* result_y, size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int sum_x = 0;
    int sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
      int x = j + k - 1;
      int y = i + k - 1;
      x = Clamp(x, 0, width - 1);
      y = Clamp(y, 0, height - 1);
      sum_y += g_y[i * width + x] * d_kernel_gradient[k];
      sum_x += g_x[y * width + j] * d_kernel_gradient[k];
    }
    result_x[i * width + j] = sum_x;
    result_y[i * width + j] = sum_y;
  }
}

__global__ void CudaSepKernelDiff(int* g_x, int* g_y, uint16_t* dst,
                                  size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int sum_x = 0;
    int sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; k++) {
      int x = j + k - 1;
      int y = i + k - 1;
      x = Clamp(x, 0, width - 1);
      y = Clamp(y, 0, height - 1);
      sum_y += g_y[i * width + x] * d_kernel_gradient[k];
      sum_x += g_x[y * width + j] * d_kernel_gradient[k];
    }
    int sum = abs(sum_x) + abs(sum_y);
    dst[i * width + j] = static_cast<uint16_t>(Clamp(sum, 0, 65535));
  }
}

__global__ void CudaSepKernelDiffShared(int* g_x, int* g_y, int* result_x,
                                        int* result_y, size_t height,
                                        size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  int li = Clamp(i, 0, height - 1);
  int lj = Clamp(j, 0, width - 1);
  __shared__ int s_tile_gx[kBlockSize + 2][kBlockSize];
  __shared__ int s_tile_gy[kBlockSize][kBlockSize + 2];
  s_tile_gx[local_y + 1][local_x] = g_x[li * width + lj];
  s_tile_gy[local_y][local_x + 1] = g_y[li * width + lj];
  if (local_y == 0) {
    s_tile_gx[0][local_x] = g_x[max(li - 1, 0) * width + lj];
    s_tile_gx[kBlockSize + 1][local_x] =
        g_x[min(li + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x == 0) {
    s_tile_gy[local_y][0] = g_y[li * width + max(lj - 1, 0)];
    s_tile_gy[local_y][kBlockSize + 1] =
        g_y[li * width + min(lj + kBlockSize, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; ++k) {
      sum_x += s_tile_gx[local_y + k][local_x] * d_kernel_gradient[k];
      sum_y += s_tile_gy[local_y][local_x + k] * d_kernel_gradient[k];
    }
    result_x[i * width + j] = sum_x;
    result_y[i * width + j] = sum_y;
  }
}

__global__ void CudaSepKernelDiffShared(int* g_x, int* g_y, uint16_t* dst,
                                        size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  int li = Clamp(i, 0, height - 1);
  int lj = Clamp(j, 0, width - 1);
  __shared__ int s_tile_gx[kBlockSize + 2][kBlockSize];
  __shared__ int s_tile_gy[kBlockSize][kBlockSize + 2];
  s_tile_gx[local_y + 1][local_x] = g_x[li * width + lj];
  s_tile_gy[local_y][local_x + 1] = g_y[li * width + lj];
  if (local_y == 0) {
    s_tile_gx[0][local_x] = g_x[max(li - 1, 0) * width + lj];
    s_tile_gx[kBlockSize + 1][local_x] =
        g_x[min(li + kBlockSize, (int)height - 1) * width + lj];
  }
  if (local_x == 0) {
    s_tile_gy[local_y][0] = g_y[li * width + max(lj - 1, 0)];
    s_tile_gy[local_y][kBlockSize + 1] =
        g_y[li * width + min(lj + kBlockSize, (int)width - 1)];
  }
  __syncthreads();
  if (i < height && j < width) {
    int sum_x = 0, sum_y = 0;
#pragma unroll
    for (int k = 0; k < 3; ++k) {
      sum_x += s_tile_gx[local_y + k][local_x] * d_kernel_gradient[k];
      sum_y += s_tile_gy[local_y][local_x + k] * d_kernel_gradient[k];
    }
    int sum = abs(sum_x) + abs(sum_y);
    dst[i * width + j] = static_cast<uint16_t>(Clamp(sum, 0, 65535));
  }
}

/**
 * @brief Складывает абсолютные значения двух матриц поэлементно.
 *
 * @param mtx1 Указатель на первую матрицу.
 * @param mtx2 Указатель на вторую матрицу.
 * @param result Указатель на результирующую матрицу.
 * @param height Высота матриц.
 * @param width Ширина матриц.
 */
__global__ void CudaAddAbsMtx(int* mtx1, int* mtx2, uint16_t* result,
                              size_t height, size_t width) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    int sum = abs(mtx1[i * width + j]) + abs(mtx2[i * width + j]);
    result[i * width + j] = static_cast<uint16_t>(Clamp(sum, 0, 65535));
  }
}

/**
 * @brief Применяет размытие по Гауссу к входному изображению.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param dst Указатель на результирующее изображение на устройстве.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 * @param kernel Указатель на ядро Гаусса на устройстве.
 * @param ksize Размер ядра (предполагается квадратное).
 */
__global__ void CudaGaussianBlur(uint16_t* src, uint16_t* dst, size_t height,
                                 size_t width, float* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    float sum = 0;
    for (int k = 0; k < ksize; k++) {
      for (int l = 0; l < ksize; l++) {
        int x = j + l - ksize / 2;
        int y = i + k - ksize / 2;
        x = Clamp(x, 0, width - 1);
        y = Clamp(y, 0, height - 1);
        sum += src[y * width + x] * kernel[k * ksize + l];
      }
    }
    dst[i * width + j] = static_cast<uint16_t>(Clamp(round(sum), 0, 65535));
  }
}

__global__ void CudaGaussianBlurShared(uint16_t* src, uint16_t* dst,
                                       size_t height, size_t width,
                                       float* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int radius = static_cast<int>(ksize) / 2;
  const int tile_w = kBlockSize + 2 * radius;
  extern __shared__ uint16_t s_tile[];

  int li = Clamp(i - radius, 0, static_cast<int>(height) - 1);
  int lj = Clamp(j - radius, 0, static_cast<int>(width) - 1);
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  s_tile[ly * tile_w + lx] = src[static_cast<size_t>(li) * width + lj];
  if (ly < 2 * radius) {
    int gy = min(i - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(height) - 1);
    s_tile[(ly + kBlockSize) * tile_w + lx] =
        src[static_cast<size_t>(gy) * width + lj];
  }
  if (lx < 2 * radius) {
    int gx = min(j - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(width) - 1);
    s_tile[ly * tile_w + (lx + kBlockSize)] =
        src[static_cast<size_t>(li) * width + gx];
  }
  if (ly >= static_cast<int>(kBlockSize) - 2 * radius &&
      lx >= static_cast<int>(kBlockSize) - 2 * radius) {
    int gy = min(i + radius, static_cast<int>(height) - 1);
    int gx = min(j + radius, static_cast<int>(width) - 1);
    s_tile[(ly + 2 * radius) * tile_w + (lx + 2 * radius)] =
        src[static_cast<size_t>(gy) * width + gx];
  }
  __syncthreads();
  if (i < height && j < width) {
    float sum = 0.0f;
    int k_int = static_cast<int>(ksize);
    for (int r = 0; r < k_int; ++r) {
      for (int c = 0; c < k_int; ++c) {
        int val = s_tile[(threadIdx.y + r) * tile_w + (threadIdx.x + c)];
        sum += static_cast<float>(val) * kernel[r * k_int + c];
      }
    }
    dst[i * width + j] = static_cast<uint16_t>(Clamp(lrintf(sum), 0, 65535));
  }
}

/**
 * @brief Применяет горизонтальное размытие по Гауссу к входному изображению.
 *
 * @param src Указатель на исходное изображение на устройстве.
 * @param dst Указатель на промежуточное изображение (результат
 * горизонтального размытия).
 * @param height Высота изображения.
 * @param width Ширина изображения.
 * @param kernel Указатель на горизонтальное ядро Гаусса на устройстве.
 * @param ksize Размер ядра.
 */
__global__ void CudaGaussianBlurSepHorizontal(uint16_t* src, float* dst,
                                              size_t height, size_t width,
                                              float* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    float sum = 0;
    for (int k = 0; k < ksize; k++) {
      int x = j + k - ksize / 2;
      x = Clamp(x, 0, width - 1);
      sum += src[i * width + x] * kernel[k];
    }
    dst[i * width + j] = sum;
  }
}

/**
 * @brief Применяет вертикальное размытие по Гауссу к промежуточному
 * изображению.
 *
 * @param src Указатель на промежуточное изображение (результат
 * горизонтального размытия).
 * @param dst Указатель на результирующее изображение на устройстве.
 * @param height Высота изображения.
 * @param width Ширина изображения.
 * @param kernel Указатель на вертикальное ядро Гаусса на устройстве.
 * @param ksize Размер ядра.
 */
__global__ void CudaGaussianBlurSepVertical(float* src, uint16_t* dst,
                                            size_t height, size_t width,
                                            float* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < height && j < width) {
    float sum = 0;
    for (int k = 0; k < ksize; k++) {
      int y = i + k - ksize / 2;
      y = Clamp(y, 0, height - 1);
      sum += src[y * width + j] * kernel[k];
    }
    dst[i * width + j] = static_cast<uint16_t>(Clamp(round(sum), 0, 65535));
  }
}

__global__ void CudaGaussianBlurSepHorizontalShared(uint16_t* src, float* dst,
                                                    size_t height, size_t width,
                                                    float* kernel,
                                                    size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int radius = static_cast<int>(ksize) / 2;
  const int tile_w = kBlockSize + 2 * radius;
  extern __shared__ uint16_t s_tile_u16[];
  int li = Clamp(i - radius, 0, static_cast<int>(height) - 1);
  int lj = Clamp(j - radius, 0, static_cast<int>(width) - 1);
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  s_tile_u16[ly * tile_w + lx] = src[static_cast<size_t>(li) * width + lj];
  if (ly < 2 * radius) {
    int gy = min(i - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(height) - 1);
    s_tile_u16[(ly + kBlockSize) * tile_w + lx] =
        src[static_cast<size_t>(gy) * width + lj];
  }
  if (lx < 2 * radius) {
    int gx = min(j - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(width) - 1);
    s_tile_u16[ly * tile_w + (lx + kBlockSize)] =
        src[static_cast<size_t>(li) * width + gx];
  }
  if (ly >= static_cast<int>(kBlockSize) - 2 * radius &&
      lx >= static_cast<int>(kBlockSize) - 2 * radius) {
    int gy = min(i + radius, static_cast<int>(height) - 1);
    int gx = min(j + radius, static_cast<int>(width) - 1);
    s_tile_u16[(ly + 2 * radius) * tile_w + (lx + 2 * radius)] =
        src[static_cast<size_t>(gy) * width + gx];
  }
  __syncthreads();
  if (i < height && j < width) {
    float sum = 0.0f;
    int k_int = static_cast<int>(ksize);
    for (int c = 0; c < k_int; ++c) {
      int val = s_tile_u16[(threadIdx.y + radius) * tile_w + (threadIdx.x + c)];
      sum += static_cast<float>(val) * kernel[c];
    }
    dst[i * width + j] = sum;
  }
}

__global__ void CudaGaussianBlurSepVerticalShared(float* src, uint16_t* dst,
                                                  size_t height, size_t width,
                                                  float* kernel, size_t ksize) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int radius = static_cast<int>(ksize) / 2;
  const int tile_w = kBlockSize + 2 * radius;
  extern __shared__ float s_tile_f32[];
  int li = Clamp(i - radius, 0, static_cast<int>(height) - 1);
  int lj = Clamp(j - radius, 0, static_cast<int>(width) - 1);
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  s_tile_f32[ly * tile_w + lx] = src[static_cast<size_t>(li) * width + lj];
  if (ly < 2 * radius) {
    int gy = min(i - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(height) - 1);
    s_tile_f32[(ly + kBlockSize) * tile_w + lx] =
        src[static_cast<size_t>(gy) * width + lj];
  }
  if (lx < 2 * radius) {
    int gx = min(j - radius + static_cast<int>(kBlockSize),
                 static_cast<int>(width) - 1);
    s_tile_f32[ly * tile_w + (lx + kBlockSize)] =
        src[static_cast<size_t>(li) * width + gx];
  }
  if (ly >= static_cast<int>(kBlockSize) - 2 * radius &&
      lx >= static_cast<int>(kBlockSize) - 2 * radius) {
    int gy = min(i + radius, static_cast<int>(height) - 1);
    int gx = min(j + radius, static_cast<int>(width) - 1);
    s_tile_f32[(ly + 2 * radius) * tile_w + (lx + 2 * radius)] =
        src[static_cast<size_t>(gy) * width + gx];
  }
  __syncthreads();
  if (i < height && j < width) {
    float sum = 0.0f;
    int k_int = static_cast<int>(ksize);
    for (int r = 0; r < k_int; ++r) {
      float val =
          s_tile_f32[(threadIdx.y + r) * tile_w + (threadIdx.x + radius)];
      sum += val * kernel[r];
    }
    dst[i * width + j] = static_cast<uint16_t>(Clamp(lrintf(sum), 0, 65535));
  }
}

TIFFImage TIFFImage::SetKernelCuda(const Kernel<int>& kernel,
                                   const bool shared_memory,
                                   const bool rotate) const {
  uint16_t* h_src = image_;
  uint16_t* d_src;
  uint16_t* h_dst = new uint16_t[width_ * height_];
  uint16_t* d_dst;
  size_t image_size = width_ * height_ * sizeof(uint16_t);
  if (!cuda_mem_manager_.IsAllocated()) {
    size_t free_memory, total_memory;
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < image_size * 2) {
      throw std::runtime_error("Изображение слишком большое для GPU");
    }
    checkCudaErrors(hipMalloc(&d_src, image_size));
    checkCudaErrors(
        hipMemcpy(d_src, h_src, image_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_dst, image_size));
  } else {
    d_src = cuda_mem_manager_.GetDeviceSrc();
    d_dst = cuda_mem_manager_.GetDeviceDst();
  }
  dim3 threads = shared_memory ? dim3(kBlockSize, kBlockSize) : dim3(1024);
  dim3 blocks = shared_memory ? dim3((width_ + kBlockSize - 1) / kBlockSize,
                                     (height_ + kBlockSize - 1) / kBlockSize)
                              : dim3((width_ + 1023) / 1024, height_);
  if (kernel == kKernelSobel) {
    if (shared_memory) {
      CudaSetSobelKernelShared<<<blocks, threads>>>(d_src, d_dst, height_,
                                                    width_);
    } else {
      CudaSetSobelKernel<<<blocks, threads>>>(d_src, d_dst, height_, width_);
    }
  } else if (kernel == kKernelPrewitt) {
    if (shared_memory) {
      CudaSetPrewittKernelShared<<<blocks, threads>>>(d_src, d_dst, height_,
                                                      width_);
    } else {
      CudaSetPrewittKernel<<<blocks, threads>>>(d_src, d_dst, height_, width_);
    }
  } else {
    int* h_kernel = new int[kernel.GetHeight() * kernel.GetWidth()];
    size_t kernel_size = kernel.GetHeight() * kernel.GetWidth() * sizeof(int);
    for (size_t i = 0; i < kernel.GetHeight(); i++) {
      for (size_t j = 0; j < kernel.GetWidth(); j++) {
        // Kernel::Get expects (x, y) i.e., (column, row)
        h_kernel[i * kernel.GetWidth() + j] = kernel.Get(j, i);
      }
    }
    int* d_kernel;
    checkCudaErrors(hipMalloc(&d_kernel, kernel_size));
    checkCudaErrors(
        hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice));
    delete[] h_kernel;
    if (shared_memory) {
      const int ksize = static_cast<int>(kernel.GetHeight());
      const int radius = ksize / 2;
      const size_t sh_w = kBlockSize + 2 * radius;
      const size_t sh_h = kBlockSize + 2 * radius;
      const size_t shared_bytes = sh_w * sh_h * sizeof(uint16_t);
      if (rotate) {
        CudaSetKernelRotateShared<<<blocks, threads, shared_bytes>>>(
            d_src, d_dst, height_, width_, d_kernel, ksize);
      } else {
        CudaSetKernelShared<<<blocks, threads, shared_bytes>>>(
            d_src, d_dst, height_, width_, d_kernel, ksize);
      }
    } else {
      if (rotate) {
        CudaSetKernelRotate<<<blocks, threads>>>(d_src, d_dst, height_, width_,
                                                 d_kernel, kernel.GetHeight());
      } else {
        CudaSetKernel<<<blocks, threads>>>(d_src, d_dst, height_, width_,
                                           d_kernel, kernel.GetHeight());
      }
    }
    checkCudaErrors(hipFree(d_kernel));
  }
  // checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_dst, d_dst, image_size, hipMemcpyDeviceToHost));
  if (!cuda_mem_manager_.IsAllocated()) {
    checkCudaErrors(hipFree(d_src));
    checkCudaErrors(hipFree(d_dst));
  }
  TIFFImage result(*this);
  std::memcpy(result.image_, h_dst, image_size);
  delete[] h_dst;
  return result;
}

TIFFImage TIFFImage::SetKernelSobelSepCuda(const bool shared_memory) const {
  uint16_t* h_src = image_;
  uint16_t* d_src;
  int* d_g_x;
  int* d_g_y;
  int* d_result_x;
  int* d_result_y;
  uint16_t* h_dst = new uint16_t[width_ * height_];
  uint16_t* d_dst;
  size_t image_size = width_ * height_ * sizeof(uint16_t);
  size_t temps_size = image_size * 2;
  if (!cuda_mem_manager_.IsAllocated()) {
    size_t free_memory, total_memory;
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < image_size * 2 + temps_size * 4) {
      throw std::runtime_error("Изображение слишком большое для GPU");
    }
    checkCudaErrors(hipMalloc(&d_src, image_size));
    checkCudaErrors(
        hipMemcpy(d_src, h_src, image_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_g_x, temps_size));
    checkCudaErrors(hipMalloc(&d_g_y, temps_size));
    checkCudaErrors(hipMalloc(&d_result_x, temps_size));
    checkCudaErrors(hipMalloc(&d_result_y, temps_size));
    checkCudaErrors(hipMalloc(&d_dst, image_size));
  } else {
    d_src = cuda_mem_manager_.GetDeviceSrc();
    d_g_x = cuda_mem_manager_.GetDeviceSepGx();
    d_g_y = cuda_mem_manager_.GetDeviceSepGy();
    d_result_x = cuda_mem_manager_.GetDeviceSepResultX();
    d_result_y = cuda_mem_manager_.GetDeviceSepResultY();
    d_dst = cuda_mem_manager_.GetDeviceDst();
  }
  dim3 threads = shared_memory ? dim3(kBlockSize, kBlockSize) : dim3(1024);
  dim3 blocks = shared_memory ? dim3((width_ + kBlockSize - 1) / kBlockSize,
                                     (height_ + kBlockSize - 1) / kBlockSize)
                              : dim3((width_ + 1023) / 1024, height_);
  if (shared_memory) {
    CudaSetSobelKernelSmoothShared<<<blocks, threads>>>(d_src, d_g_x, d_g_y,
                                                        height_, width_);
    // checkCudaErrors(hipDeviceSynchronize());
    CudaSepKernelDiffShared<<<blocks, threads>>>(d_g_x, d_g_y, d_dst, height_,
                                                 width_);
    // CudaSepKernelDiffShared<<<blocks, threads>>>(d_g_x, d_g_y, d_result_x,
    //                                              d_result_y, height_,
    //                                              width_);
    // checkCudaErrors(hipDeviceSynchronize());
    // CudaAddAbsMtx<<<blocks, threads>>>(d_result_x, d_result_y, d_dst,
    // height_,
    //                                    width_);
    // checkCudaErrors(hipDeviceSynchronize());
  } else {
    CudaSetSobelKernelSmooth<<<blocks, threads>>>(d_src, d_g_x, d_g_y, height_,
                                                  width_);
    // checkCudaErrors(hipDeviceSynchronize());
    CudaSepKernelDiff<<<blocks, threads>>>(d_g_x, d_g_y, d_dst, height_,
                                           width_);
    // CudaSepKernelDiff<<<blocks, threads>>>(d_g_x, d_g_y, d_result_x,
    // d_result_y,
    //                                        height_, width_);
    // checkCudaErrors(hipDeviceSynchronize());
    // CudaAddAbsMtx<<<blocks, threads>>>(d_result_x, d_result_y, d_dst,
    // height_,
    //                                    width_);
    // checkCudaErrors(hipDeviceSynchronize());
  }
  checkCudaErrors(hipMemcpy(h_dst, d_dst, image_size, hipMemcpyDeviceToHost));
  if (!cuda_mem_manager_.IsAllocated()) {
    checkCudaErrors(hipFree(d_src));
    checkCudaErrors(hipFree(d_g_x));
    checkCudaErrors(hipFree(d_g_y));
    checkCudaErrors(hipFree(d_result_x));
    checkCudaErrors(hipFree(d_result_y));
    checkCudaErrors(hipFree(d_dst));
  }
  TIFFImage result(*this);
  std::memcpy(result.image_, h_dst, image_size);
  delete[] h_dst;
  return result;
}

TIFFImage TIFFImage::SetKernelPrewittSepCuda(const bool shared_memory) const {
  uint16_t* h_src = image_;
  uint16_t* d_src;
  int* d_g_x;
  int* d_g_y;
  int* d_result_x;
  int* d_result_y;
  uint16_t* h_dst = new uint16_t[width_ * height_];
  uint16_t* d_dst;
  size_t image_size = width_ * height_ * sizeof(uint16_t);
  size_t temps_size = image_size * 2;
  if (!cuda_mem_manager_.IsAllocated()) {
    size_t free_memory, total_memory;
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < image_size * 2 + temps_size * 4) {
      throw std::runtime_error("Изображение слишком большое для GPU");
    }
    checkCudaErrors(hipMalloc(&d_src, image_size));
    checkCudaErrors(
        hipMemcpy(d_src, h_src, image_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_g_x, temps_size));
    checkCudaErrors(hipMalloc(&d_g_y, temps_size));
    checkCudaErrors(hipMalloc(&d_result_x, temps_size));
    checkCudaErrors(hipMalloc(&d_result_y, temps_size));
    checkCudaErrors(hipMalloc(&d_dst, image_size));
  } else {
    d_src = cuda_mem_manager_.GetDeviceSrc();
    d_g_x = cuda_mem_manager_.GetDeviceSepGx();
    d_g_y = cuda_mem_manager_.GetDeviceSepGy();
    d_result_x = cuda_mem_manager_.GetDeviceSepResultX();
    d_result_y = cuda_mem_manager_.GetDeviceSepResultY();
    d_dst = cuda_mem_manager_.GetDeviceDst();
  }
  dim3 threads = shared_memory ? dim3(kBlockSize, kBlockSize) : dim3(1024);
  dim3 blocks = shared_memory ? dim3((width_ + kBlockSize - 1) / kBlockSize,
                                     (height_ + kBlockSize - 1) / kBlockSize)
                              : dim3((width_ + 1023) / 1024, height_);
  if (shared_memory) {
    CudaSetPrewittKernelAverageShared<<<blocks, threads>>>(d_src, d_g_x, d_g_y,
                                                           height_, width_);
    // checkCudaErrors(hipDeviceSynchronize());
    CudaSepKernelDiffShared<<<blocks, threads>>>(d_g_x, d_g_y, d_dst, height_,
                                                 width_);
    // CudaSepKernelDiffShared<<<blocks, threads>>>(d_g_x, d_g_y, d_result_x,
    //                                              d_result_y, height_,
    //                                              width_);
    // checkCudaErrors(hipDeviceSynchronize());
    // CudaAddAbsMtx<<<blocks, threads>>>(d_result_x, d_result_y, d_dst,
    // height_,
    //                                    width_);
    // checkCudaErrors(hipDeviceSynchronize());
  } else {
    CudaSetPrewittKernelAverage<<<blocks, threads>>>(d_src, d_g_x, d_g_y,
                                                     height_, width_);
    // checkCudaErrors(hipDeviceSynchronize());
    CudaSepKernelDiff<<<blocks, threads>>>(d_g_x, d_g_y, d_dst, height_,
                                           width_);
    // CudaSepKernelDiff<<<blocks, threads>>>(d_g_x, d_g_y, d_result_x,
    // d_result_y,
    //                                        height_, width_);
    // checkCudaErrors(hipDeviceSynchronize());
    // CudaAddAbsMtx<<<blocks, threads>>>(d_result_x, d_result_y, d_dst,
    // height_,
    //                                    width_);
    // checkCudaErrors(hipDeviceSynchronize());
  }
  checkCudaErrors(hipMemcpy(h_dst, d_dst, image_size, hipMemcpyDeviceToHost));
  if (!cuda_mem_manager_.IsAllocated()) {
    checkCudaErrors(hipFree(d_src));
    checkCudaErrors(hipFree(d_g_x));
    checkCudaErrors(hipFree(d_g_y));
    checkCudaErrors(hipFree(d_result_x));
    checkCudaErrors(hipFree(d_result_y));
    checkCudaErrors(hipFree(d_dst));
  }
  TIFFImage result(*this);
  std::memcpy(result.image_, h_dst, image_size);
  delete[] h_dst;
  return result;
}

TIFFImage TIFFImage::GaussianBlurCuda(const size_t size, const float sigma,
                                      const bool shared_memory) {
  uint16_t* h_src = image_;
  uint16_t* d_src;
  uint16_t* h_dst = new uint16_t[width_ * height_];
  uint16_t* d_dst;
  float* d_kernel;
  size_t image_size = width_ * height_ * sizeof(uint16_t);
  if (!cuda_mem_manager_.IsAllocated()) {
    size_t free_memory, total_memory;
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < image_size * 2) {
      throw std::runtime_error("Изображение слишком большое для GPU");
    }
    checkCudaErrors(hipMalloc(&d_src, image_size));
    checkCudaErrors(
        hipMemcpy(d_src, h_src, image_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_dst, image_size));
    Kernel<float> kernel = Kernel<float>::GetGaussianKernel(size, sigma);
    float* h_kernel;
    size_t kernel_size = kernel.GetHeight() * kernel.GetWidth() * sizeof(float);
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < kernel_size) {
      throw std::runtime_error("Ядро слишком большое для GPU");
    }
    kernel.CopyKernelTo(&h_kernel);
    checkCudaErrors(hipMalloc(&d_kernel, kernel_size));
    checkCudaErrors(
        hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice));
    delete[] h_kernel;
  } else {
    d_src = cuda_mem_manager_.GetDeviceSrc();
    d_dst = cuda_mem_manager_.GetDeviceDst();
    cuda_mem_manager_.CheckGaussianKernel(size, sigma);
    d_kernel = cuda_mem_manager_.GetDeviceGaussianKernel();
  }
  if (shared_memory) {
    dim3 threads(kBlockSize, kBlockSize);
    dim3 blocks((width_ + kBlockSize - 1) / kBlockSize,
                (height_ + kBlockSize - 1) / kBlockSize);
    const int radius = static_cast<int>(size) / 2;
    const size_t sh_w = kBlockSize + 2 * radius;
    const size_t sh_h = kBlockSize + 2 * radius;
    const size_t shared_bytes = sh_w * sh_h * sizeof(uint16_t);
    CudaGaussianBlurShared<<<blocks, threads, shared_bytes>>>(
        d_src, d_dst, height_, width_, d_kernel, size);
  } else {
    dim3 threads(1024);
    dim3 blocks((width_ + 1023) / 1024, height_);
    CudaGaussianBlur<<<blocks, threads>>>(d_src, d_dst, height_, width_,
                                          d_kernel, size);
  }
  // checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_dst, d_dst, image_size, hipMemcpyDeviceToHost));
  if (!cuda_mem_manager_.IsAllocated()) {
    checkCudaErrors(hipFree(d_src));
    checkCudaErrors(hipFree(d_dst));
    checkCudaErrors(hipFree(d_kernel));
  }
  TIFFImage result(*this);
  std::memcpy(result.image_, h_dst, image_size);
  delete[] h_dst;
  return result;
}

TIFFImage TIFFImage::GaussianBlurSepCuda(const size_t size, const float sigma,
                                         const bool shared_memory) {
  uint16_t* h_src = image_;
  uint16_t* d_src;
  float* d_temp;
  uint16_t* h_dst = new uint16_t[width_ * height_];
  uint16_t* d_dst;
  float* d_kernel;
  size_t image_size = width_ * height_ * sizeof(uint16_t);
  size_t temp_size = width_ * height_ * sizeof(float);
  if (!cuda_mem_manager_.IsAllocated()) {
    size_t free_memory, total_memory;
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < image_size * 2 + temp_size) {
      throw std::runtime_error("Изображение слишком большое для GPU");
    }
    checkCudaErrors(hipMalloc(&d_src, image_size));
    checkCudaErrors(
        hipMemcpy(d_src, h_src, image_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&d_temp, temp_size));
    checkCudaErrors(hipMalloc(&d_dst, image_size));
    Kernel<float> kernel = Kernel<float>::GetGaussianKernelSep(size, sigma);
    float* h_kernel;
    size_t kernel_size = kernel.GetHeight() * kernel.GetWidth() * sizeof(float);
    checkCudaErrors(hipMemGetInfo(&free_memory, &total_memory));
    if (free_memory < kernel_size) {
      throw std::runtime_error("Ядро слишком большое для GPU");
    }
    kernel.CopyKernelTo(&h_kernel);
    checkCudaErrors(hipMalloc(&d_kernel, kernel_size));
    checkCudaErrors(
        hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice));
    delete[] h_kernel;
  } else {
    d_src = cuda_mem_manager_.GetDeviceSrc();
    d_temp = cuda_mem_manager_.GetDeviceGaussianSepTemp();
    d_dst = cuda_mem_manager_.GetDeviceDst();
    cuda_mem_manager_.CheckGaussianKernel(size, sigma);
    d_kernel = cuda_mem_manager_.GetDeviceGaussianKernel();
  }
  if (shared_memory) {
    dim3 threads(kBlockSize, kBlockSize);
    dim3 blocks((width_ + kBlockSize - 1) / kBlockSize,
                (height_ + kBlockSize - 1) / kBlockSize);
    const int radius = static_cast<int>(size) / 2;
    const size_t sh_w = kBlockSize + 2 * radius;
    const size_t sh_h = kBlockSize + 2 * radius;
    size_t shared_bytes_h = sh_w * sh_h * sizeof(uint16_t);
    size_t shared_bytes_v = sh_w * sh_h * sizeof(float);
    CudaGaussianBlurSepHorizontalShared<<<blocks, threads, shared_bytes_h>>>(
        d_src, d_temp, height_, width_, d_kernel, size);
    CudaGaussianBlurSepVerticalShared<<<blocks, threads, shared_bytes_v>>>(
        d_temp, d_dst, height_, width_, d_kernel, size);
  } else {
    dim3 threads(1024);
    dim3 blocks((width_ + 1023) / 1024, height_);
    CudaGaussianBlurSepHorizontal<<<blocks, threads>>>(d_src, d_temp, height_,
                                                       width_, d_kernel, size);
    CudaGaussianBlurSepVertical<<<blocks, threads>>>(d_temp, d_dst, height_,
                                                     width_, d_kernel, size);
  }
  // checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_dst, d_dst, image_size, hipMemcpyDeviceToHost));
  if (!cuda_mem_manager_.IsAllocated()) {
    checkCudaErrors(hipFree(d_src));
    checkCudaErrors(hipFree(d_temp));
    checkCudaErrors(hipFree(d_dst));
    checkCudaErrors(hipFree(d_kernel));
  }
  TIFFImage result(*this);
  std::memcpy(result.image_, h_dst, image_size);
  delete[] h_dst;
  return result;
}